#include "hip/hip_runtime.h"
#include "simrl/tensor.hpp"
#include "simrl/utils/logging.hpp"
#include <hip/hip_runtime.h>

using namespace simrl;

constexpr size_t DIM_0 = 6;
constexpr size_t DIM_1 = 4;
constexpr size_t NUM_ELEMENTS = DIM_0 * DIM_1;

__global__ void validate_zero_kernel(float* data, size_t numel, bool* result) {
    size_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < numel && data[i] != 0.0f) {
        *result = false;
    }
}

void test_zero_cuda() {
    SIMRL_INFO("Testing zero() on CUDA tensor...");
    Tensor tensor({DIM_0, DIM_1}, DType::Float32, DeviceType::CUDA);
    tensor.zero();

    bool* device_result;
    bool host_result = true;
    SIMRL_CHECK(hipMalloc(&device_result, sizeof(bool)));
    SIMRL_CHECK(hipMemcpy(device_result, &host_result, sizeof(bool), hipMemcpyHostToDevice));

    validate_zero_kernel<<<(NUM_ELEMENTS + 31) / 32, 32>>>(
        tensor.as<float>(), NUM_ELEMENTS, device_result
    );
    SIMRL_CHECK(hipGetLastError());  // Check for kernel launch errors

    SIMRL_CHECK(hipMemcpy(&host_result, device_result, sizeof(bool), hipMemcpyDeviceToHost));
    SIMRL_CHECK(hipFree(device_result));

    SIMRL_ASSERT(host_result, "CUDA zero() failed");
}

void test_copy_from_cuda() {
    SIMRL_INFO("Testing copy_from() on CUDA tensor...");
    Tensor src({DIM_0, DIM_1}, DType::Float32, DeviceType::CUDA);
    Tensor dst({DIM_0, DIM_1}, DType::Float32, DeviceType::CUDA);
    src.zero();
    dst.copy_from(src);

    bool* device_result;
    bool host_result = true;
    SIMRL_CHECK(hipMalloc(&device_result, sizeof(bool)));
    SIMRL_CHECK(hipMemcpy(device_result, &host_result, sizeof(bool), hipMemcpyHostToDevice));

    validate_zero_kernel<<<(NUM_ELEMENTS + 31) / 32, 32>>>(
        dst.as<float>(), NUM_ELEMENTS, device_result
    );
    SIMRL_CHECK(hipGetLastError());

    SIMRL_CHECK(hipMemcpy(&host_result, device_result, sizeof(bool), hipMemcpyDeviceToHost));
    SIMRL_CHECK(hipFree(device_result));

    SIMRL_ASSERT(host_result, "CUDA copy_from() failed");
}

// test tensor clone functionality
void test_clone_cuda() {
    SIMRL_INFO("Testing clone() on CUDA tensor...");
    Tensor original({DIM_0, DIM_1}, DType::Float32, DeviceType::CUDA);
    original.zero();
    Tensor clone = original.clone();

    SIMRL_ASSERT(clone.is_cuda(), "Clone should be on CUDA device");
    SIMRL_ASSERT(clone.shape() == original.shape(), "Clone shape mismatch");
    SIMRL_ASSERT(clone.dtype() == original.dtype(), "Clone dtype mismatch");
    SIMRL_ASSERT(clone.device() == original.device(), "Clone device mismatch");

    bool* device_result;
    bool host_result = true;
    
    SIMRL_CHECK(hipMalloc(&device_result, sizeof(bool)));
    SIMRL_CHECK(hipMemcpy(device_result, &host_result, sizeof(bool), hipMemcpyHostToDevice));
    validate_zero_kernel<<<(NUM_ELEMENTS + 31) / 32, 32>>>(
        clone.as<float>(), NUM_ELEMENTS, device_result
    );
    SIMRL_CHECK(hipGetLastError());
    SIMRL_CHECK(hipMemcpy(&host_result, device_result, sizeof(bool), hipMemcpyDeviceToHost));
    SIMRL_CHECK(hipFree(device_result));

    SIMRL_ASSERT(host_result, "CUDA clone() failed");
}

auto main() -> int {
    try {
        test_zero_cuda();
        test_copy_from_cuda();
        SIMRL_INFO("✅ All CUDA tensor utility tests passed.");
    } catch (const std::exception& e) {
        SIMRL_ERROR(e.what());
        return 1;
    }
    return 0;
}
